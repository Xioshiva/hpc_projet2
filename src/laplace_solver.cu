
#include <hip/hip_runtime.h>
#include <cstdio>

#define gpuErrchk(ans)                    \
  {                                       \
    gpuAssert((ans), __FILE__, __LINE__); \
  }

inline void gpuAssert(
    hipError_t code,
    const char *file,
    int line)
{
  if (code != hipSuccess)
  {
    fprintf(
        stderr, "GPUassert: %s %s %d\n",
        hipGetErrorString(code), file, line);
  }
}

__global__ void dummy_kernel(float *domain, float *res, int N, int M)
{
  int rows = blockIdx.y * blockDim.y + threadIdx.y;
  int cols = blockIdx.x * blockDim.x + threadIdx.x;

  if (rows > 0 && cols > 0 && rows < N - 1 && cols < M - 1)
  {
    res[rows * M + cols] = 0.25 * (domain[rows * M + cols - 1] + domain[rows * M + cols + 1] + domain[(rows - 1) * M + cols] + domain[(rows + 1) * M + cols]);
  }
}

void printMat(float *mat, int M, int N)
{
  for (int i = 0; i < M * N; i++)
  {
    printf("%.3f ", mat[i]);
    if (!((i + 1) % M) && i != 0)
      printf("\n");
  }
}

// NE CHANGEZ PAS CETTE SIGNATURE ET NE DEPLACEZ PAS CETTE FONCTION !!!
float *heat_solver(
    int N, int M, int T,
    float *north, float *south,
    float *east, float *west,
    dim3 grid_dim, dim3 block_dim)
{
  if (!T)
    T = 1;
  const size_t n_bytes = sizeof(float) * (M * N);
  float *matrix = (float *)malloc(sizeof(float) * (N * M));
  for (int i = 0; i < M; i++)
  {
    matrix[i] = north[i];
    matrix[(N - 1) * M + i] = south[i];
  }
  for (int i = 0; i < N - 2; i++)
  {
    matrix[(i + 1) * M] = west[i];
    matrix[(i + 2) * M - 1] = east[i];
  }

  float *res = (float *)malloc(sizeof(float) * (N * M));
  for (int i = 0; i < M; i++)
  {
    res[i] = north[i];
    res[(N - 1) * M + i] = south[i];
  }
  for (int i = 0; i < N - 2; i++)
  {
    res[(i + 1) * M] = west[i];
    res[(i + 2) * M - 1] = east[i];
  }
  float *d_domain;
  float *d_res;
  gpuErrchk(hipMalloc(&d_domain, n_bytes));
  gpuErrchk(hipMalloc(&d_res, n_bytes));
  gpuErrchk(hipMemcpy(d_domain, matrix, n_bytes, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_res, res, n_bytes, hipMemcpyHostToDevice));

  hipEvent_t start, stop;
  gpuErrchk(hipEventCreate(&start));
  gpuErrchk(hipEventCreate(&stop));

  hipEventRecord(start);
  for (int i = 0; i < T; i++)
  {
    dummy_kernel<<<grid_dim, block_dim>>>(d_domain, d_res, N, M);
    gpuErrchk(hipDeviceSynchronize());
    float *tmp = d_res;
    d_res = d_domain;
    d_domain = tmp;
  }
  gpuErrchk(hipEventRecord(stop));
  gpuErrchk(hipEventSynchronize(stop));
  float milliseconds = 0;
  gpuErrchk(hipEventElapsedTime(&milliseconds, start, stop));
  printf("T=%d N=%d M=%d %f ms\n", T, N, M, milliseconds);
  gpuErrchk(hipMemcpy(matrix, d_domain, n_bytes, hipMemcpyDeviceToHost));
  // printMat(matrix,M, N);
  gpuErrchk(hipFree(d_domain));
  gpuErrchk(hipFree(d_res));
  free(res);
  return matrix;
}
